#include "hip/hip_runtime.h"


#include "cmovmf.hpp"
#include "cstdlib"
//#include "bessel.cpp"
#include "newbessel.h"

#define PI to_RR("3.1415926535897932384626433832795028841971693993751058209")
#define dPI to_float(PI)

__global__ void cuda_expectation_soft(float * d_alpha, float * d_c, float * d_kappa, float * d_mu, float * d_x, float * d_p, int num_clusters, int dimension, float * d_clust, float *d_f)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	float temp_sum = 0.0;
	for(int h=0; h<num_clusters; h++)
	{
		float dot_product = 0.0;
	//	float * f;

		for(int l=0; l<dimension; l++)
			dot_product += d_mu[h*dimension + l] * d_x[index * dimension + l];

		d_f[index * num_clusters + h] = d_c[h] + log(d_alpha[h]) + d_kappa[h] * dot_product;

	//	d_f[index * num_clusters + h] = index * num_clusters + h;
//		temp_sum += exp(d_f[index * num_clusters + h]);
	}
/*
	for(int h=0; h<num_clusters; h++)

		d_p[index*num_clusters + h] = d_f[index * num_clusters + h] - log(temp_sum);

	float temp_max  = d_f[index * num_clusters + 0];
	d_clust[index] = 0;
	for(int j=1;j<num_clusters;j++)
		if(temp_max < d_f[index * num_clusters + j])
		{
			temp_max = d_f[index * num_clusters + j];
			d_clust[index] = j;
		}
*/
//	__syncthreads();
}

movmf::movmf(int dim, int clusters, int vectors, float ** data)
{
	d = dim;
	k = clusters;
	n = vectors;

	data_points = data;

	alpha = new float [clusters];
	kappa = new float [clusters];

	//data_points = new float *[vectors];
	f = new float *[vectors];
	p   = new float *[vectors];
	for(int i=0;i<vectors; i++)
	{
	//	data_points[i] = new float[dim];
		f[i] = new float[clusters];
		p[i]   = new float[clusters];
	}	

	mu   = new float *[clusters];
	for(int i=0;i<clusters; i++)
		mu[i] = new float[dim];
	
	c = new float [clusters];
	//float ** r;
	clust = new int[vectors];

}

void movmf::initialize()
{
/*	alpha[0] = 0.3;
	alpha[1] = 0.7;

	mu[0][0] = 0; 
	mu[0][1] = 0.5;
	mu[0][2] = 0.5*sqrt(3);
	mu[1][0] = 0.5;
	mu[1][1] = 0.5*sqrt(3);
	mu[1][2] = 0;

	kappa[0] = 3;
	kappa[1] = 3;
	*/
	initialRandMeans();
	meansFromSpkmeans();

	float * clust_size = new float[k];
	for(int i=0;i<k;i++)
	{
		kappa[i] = 1;
		clust_size[i] = 0;
	}
	for(int i=0;i<n;i++)
		clust_size[clust[i]]++;

	for(int i=0;i<k;i++)
		alpha[i] = clust_size[i]/n;

	std::cout << "Kappa : ";
	printVec(kappa,k);
	std::cout << "Alpha : ";
	printVec(alpha,k);
	std::cout << "Mu : ";
	printMat(mu,k,d);

	//zero clust
	for(int i=0;i<n;i++)
		clust[i] = 0.0;
}


// Getting the initial random means
void movmf::initialRandMeans()
{
	// getting global mean
	float * sumv = new float[n];

	for(int j=0;j<d;j++)
		sumv[j] = 0.0;

	for(int j=0;j<d;j++)
		for(int i=0;i<n;i++)
			sumv[j] += data_points[i][j]; 

//	printVec(sumv, d);
	float * mu_0 = normalize(sumv);
//	printVec(mu_0, d);

	// perturbing global mean to get initial cluster centroids
	float perturb = 0.1;

	for(int h=0; h<k;h++)
	{
		float * randVec = new float[n];
		for(int i=0;i<n;i++)
			randVec[i] = ((float)rand()/(float)RAND_MAX) - 0.5;
		
		float randNorm = perturb * ((float)rand()/(float)RAND_MAX);

		float * smallRandVec = normalize(randVec);
		for(int i=0;i<d;i++)
		{
			smallRandVec[i] *= randNorm;
			mu[h][i] = mu_0[i] + randNorm*smallRandVec[i];		
		}	
		mu[h] = normalize(mu[h]);
		delete[] randVec;
	}

//	printMat(mu,k,d);
	delete[] sumv;
}

//Getting the means from spkmeans
void movmf::meansFromSpkmeans()
{
	float diff    = 1;
	float epsilon = 0.001;
	float value   = 100;
	int iteration = 1;

	float ** simMat = new float*[n];
	for(int i=0;i<n;i++)
	{
		simMat[i] = new float[k];
		for(int h=0;h<k;h++)
			simMat[i][h] = 0.0;
	}

	while (diff > epsilon && iteration < 10)
	{  
	//	display(['Iteration ',num2str(iteration)]);
		std::cout << "Iteration " << iteration << " ";
		std::cout << "diff = " << diff << " " << std::endl;

		iteration++;
		float oldvalue = value;

		for(int i=0;i<n;i++)
			for(int h=0;h<k;h++)
				simMat[i][h] = 0.0;
		// assign points to nearest cluster
		for(int i= 0;i<n;i++)
			for(int h=0;h<k;h++)
			{
				for(int j=0;j<d;j++)
					simMat[i][h] += data_points[i][j]*mu[h][j];
			}

	//	printMat(simMat,10,k);	

		float * simax = new float[n];
	//	int * clust = new int[n];
			
		for(int i=0;i<n;i++)
		{
			float temp_max = simMat[i][0];
			clust[i] = 0;
			for(int j=1;j<k;j++)
				if(temp_max < simMat[i][j])
				{
					temp_max = simMat[i][j];
					clust[i] = j;
				}	
			simax[i] = temp_max;	
		}	

		// compute objective function value
		value =0.0;
		for(int i=0;i<n;i++)
			value += simax[i];

		// compute cluster centroids
		for(int h=0;h<k;h++)
		{
			float * sumVec = new float[d];
			for(int j=0;j<d;j++)
				sumVec[j] = 0.0;

			for(int i=0;i<n;i++)
			{
				if(clust[i] == h)
				{
					for(int j=0;j<d;j++)
						sumVec[j] += data_points[i][j];
				}
			}	

			mu[h] = normalize(sumVec);
			delete[] sumVec;
		}
		printMat(mu, k, d);

		diff = abs(value - oldvalue);

		std::cout << "Value = " << value << " oldvalue = " << oldvalue << std::endl;

		

		delete[] simax;
	//	delete[] clust;
	}
	
	printVec(clust,n);
//	//display(clust);
//	figure;
//	subplot(2,1,1),plot(1:D,clust,'bo');
//	display('Initial iterations done');

//	Clust1 = clust;
}

float * movmf::normalize(float * vector)
{
	float * normalized = new float[d];

	float temp = 0.0;

	for(int i=0;i<d;i++)
		temp += vector[i]*vector[i];

	temp = sqrt(temp);
	for(int i=0;i<d;i++)
		normalized[i] = vector[i]/temp;

	return normalized;
}

void movmf::run()
{	
	std::cout << "****************************    CUDA part       *************************************  " << std::endl;
	float diff      = 1;
	float epsilon   = 0.0001;
	float value     = 100;
	int iteration = 1;

	printVec(clust,n);

	// float* d_c, d_x, d_mu, d_kappa, d_p, d_alpha;
	// float* h_x, h_mu, h_p;

	h_x = (float *) malloc (n*d*sizeof(float));
	h_mu = (float *) malloc (k*d*sizeof(float));
	h_p = (float *) malloc (n*k*sizeof(float));
	h_f = (float *) malloc (n*k*sizeof(float));

	hipMalloc((void**)&d_c, k*sizeof(float));
	hipMalloc((void**)&d_x, n*d*sizeof(float));
	hipMalloc((void**)&d_mu, k*d*sizeof(float));
	hipMalloc((void**)&d_kappa, k*sizeof(float));
	hipMalloc((void**)&d_p, n*k*sizeof(float));
	hipMalloc((void**)&d_alpha, n*k*sizeof(float));
	hipMalloc((void**)&d_clust, n*sizeof(float));
	hipMalloc((void**)&d_f, n*k*sizeof(float));

	


	while(diff > epsilon && iteration < 20)
	{
		std::cout << "Iteration " << iteration << " ";
		
		
		iteration++;
		float oldvalue = value;

		expectation_soft();
		//expectation_hard();
		
		value = 0.0;
		for(int i=0;i<n;i++)
			for(int h=0;h<k;h++)
				value += f[i][h];

		maximization();

		diff = abs(value - oldvalue);
		std::cout << "diff = " << diff << " " << std::endl;
		std::cout << "Value = " << value << " oldvalue = " << oldvalue << std::endl;
	}
	//printMat(p,n,k);

	printVec(clust,n);
}

void movmf::expectation_soft()
{
	calculate_norm_const();

	for(int i=0;i<n;i++)
		for(int j=0;j<d;j++)
			h_x[i*d+j] = data_points[i][j];

	for(int h=0;h<k;h++)
		for(int j=0;j<d;j++)
			h_mu[h*d+j] = mu[h][j];
		
	for(int i=0;i<n;i++)
		for(int h=0;h<k;h++)
			h_p[i*k+h] = p[i][h];

	hipMemcpy(d_alpha, alpha, k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_kappa, kappa, k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, h_x, n*d*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_mu, h_mu, k*d*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_p, h_p, n*k*sizeof(float), hipMemcpyHostToDevice);

	int THREADS_PER_BLOCK = 1;//00;
	int BLOCKS = n;// /THREADS_PER_BLOCK;

	cuda_expectation_soft <<<BLOCKS,THREADS_PER_BLOCK>>> (d_alpha, d_c, d_kappa, d_mu, d_x, d_p, k, d, d_clust, d_f);

	hipMemcpy(alpha, d_alpha, k*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(c, d_c, k*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(kappa, d_kappa, k*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_x, d_x, n*d*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_mu, d_mu, k*d*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_p, d_p, n*k*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_f, d_f, n*k*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(clust, d_clust, n*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++)
		for(int j=0;j<d;j++)
			data_points[i][j] = h_x[i*d+j];

	for(int h=0;h<k;h++)
		for(int j=0;j<d;j++)
			mu[h][j] = h_mu[h*d+j];
		
	for(int i=0;i<n;i++)
		for(int h=0;h<k;h++)
		{
			p[i][h] = h_p[i*k+h];
			f[i][h] = h_f[i*k+h];
		}	

	std::cout << " DEBUG ::: f from kernel = ";
	for(int i=0;i<n;i++)
		for(int h=0;h<k;h++)
			std::cout << h_f[i*k+h] << " ";
	std::cout <<std::endl;	

	for (int i=0; i<n; i++)
	{
	//	for(int h=0; h<k; h++)
	//		f[i][h] = c[h] + log(alpha[h]) + mult(h,i);

		float temp = 0.0;
		for(int h=0; h<k; h++)
			temp += exp(f[i][h]);

		for(int h=0; h<k; h++)
			p[i][h] = f[i][h] - log(temp);

		float temp_max  = f[i][0];
		clust[i] = 0;
		for(int j=1;j<k;j++)
			if(temp_max < f[i][j])
			{
				temp_max = f[i][j];
				clust[i] = j;
			}
	}	
}

void movmf::expectation_hard()
{
	calculate_norm_const();

	for (int i=0; i<n; i++)
	{
		for(int h=0; h<k; h++)
			f[i][h] = c[h] + log(alpha[h]) + mult(h,i);

		for(int h=0; h<k; h++)
			p[i][h] = 0.0;

		float temp_max  = f[i][0];
		clust[i] = 0;
		for(int j=1;j<k;j++)
			if(temp_max < f[i][j])
			{
				temp_max = f[i][j];
				clust[i] = j;
			}	
				
		p[i][clust[i]] = 1.0;
	}	
}

void movmf::maximization()
{
	for(int h=0; h<k; h++)
	{
		//alpha update
		alpha[h] = 0.0;
		for(int i=0; i<n; i++)
			alpha[h] += exp(p[i][h]);
		alpha[h] = alpha[h]/n;
		//mu update
		for(int j=0; j<d; j++)
		{
			mu[h][j] = 0.0;
			for(int i=0; i<n; i++)
				mu[h][j] += data_points[i][j] * exp(p[i][h]); 
		}
		//r bar
		float norm_mu = 0.0;
		for(int j=0; j<d; j++)
			norm_mu += mu[h][j] * mu[h][j];
		float r_bar = sqrt(norm_mu)/(n * alpha[h]);
		//mu update
		//normalize(mu[h]);
		for(int j=0; j<d; j++)
			mu[h][j] /= sqrt(norm_mu);

		//kappa update
		kappa[h] = (r_bar * d - r_bar * r_bar * r_bar) / (1 - r_bar * r_bar);
	}
}

float movmf::mult(int h, int i)
{
	float result = 0.0;

	for(int l=0; l<d; l++)
		result += mu[h][l] * data_points[i][l];

	result *= kappa[h];

	return result;
}

void movmf::calculate_norm_const()
{
	RR::SetPrecision(300);
  	RR bessel, nu, x;
  	float dim = d; 
//  	std::cout << "half_dim = " << dim/2 <<std::endl;
  	nu = dim/2 -1;

//  	std::cout << " C(k): ";
	for(int h=0; h<k; h++)
	{
		c[h] = 0.0;

		x = kappa[h];
	//	std::cout << " x = " << x;
	//	std::cout << " nu = " << nu;

		bessel = BesselI(nu, x);
	//	std::cout << " bessel = " << bessel;		
		
		float logBessel = to_float(log(bessel));
	
		c[h] = (d/2-1)*log(kappa[h]) - (d/2) * log(2*dPI) - logBessel;
	
	//	std::cout << " c[h] = " << c[h];
	}	 
//	std::cout << std::endl;
}

void movmf::print()
{
	std::cout << "Clustering via mixture of von Mises-Fisher";
	std::cout << std::endl << "Alpha : ";
	for(int i=0; i<k;i++)
	{
		std::cout << alpha[i] << " ";
	}

	std::cout << std::endl << "Kappa : ";
	for(int i=0; i<k;i++)
	{
		std::cout << kappa[i] << " ";
	}

	std::cout << std::endl << "Mu : ";
	for(int i=0; i<k;i++)
	{	
		std::cout << std::endl << "i = " << i << " : ";
		for(int j=0;j<d;j++)
			std::cout << mu[i][j] << " ";
	}

	std::cout << std::endl << "Posterior : ";
	for(int h=0; h<k;h++)
	{	
		std::cout << std::endl << "k = " << h << " : ";
		for(int i=0;i<n && i<10;i++)
			std::cout << p[i][h] << " ";
	}
	std::cout << std::endl;
}

void movmf::printVec(float * vec, int size)
{
	std::cout << " Vector printing : "; 
	for(int i=0;i<size;i++)
		std::cout << vec[i] << " ";
	std::cout << std::endl;
}

void movmf::printVec(int * vec, int size)
{
	std::cout << " Vector printing : "; 
	for(int i=0;i<size;i++)
		std::cout << vec[i] << " ";
	std::cout << std::endl;
}

void movmf::printMat(float ** mat, int size1, int size2)
{
	std::cout << " Matrix printing : " << std::endl; 
	for(int i=0; i<size1;i++)
	{	
		//std::cout << std::endl << "i = " << i << " : ";
		for(int j=0;j<size2;j++)
			std::cout << mat[i][j] << " ";
		std::cout << std::endl;
	}
}



int main(int argc, char **argv)
{
	std::string filename = "dataset.dat";
	std::ifstream dataStream;
	dataStream.open(filename.c_str(), std::ios::in); // open data file
	if (!dataStream) {
		std::cerr << "Cannot open data file\n";
		exit(1);
	}
	std::istream * dataIn = &dataStream;				// make this the data stream

	int dimension;
	*dataIn >> dimension;
	
	int size;
	*dataIn >> size;

	float ** data = new float *[size];
	for(int i=0;i<size;i++)
	{	
		data[i] = new float[dimension];
		for(int j=0;j<dimension;j++)
			*dataIn >> data[i][j];
	}

	std::cout << "Size = " << size << std::endl;
	std::cout << "Dimension = " << dimension << std::endl;
	for(int i=0;i<size && i<10;i++)
	{	
		std::cout << "data[" << i << "] = "; 
		for(int j=0; j<dimension; j++)
			std::cout << data[i][j] << " ";
		std::cout << std::endl;
	}	
	
	for(int i=0;i<size;i++) //normalizing the data
	{
		float sqsum = 0;
		for(int j=0;j<dimension;j++)
			sqsum += data[i][j]*data[i][j];
		float norm = sqrt(sqsum);
		for(int j=0;j<dimension;j++)
			data[i][j] /= norm;
	}
/*
	for(int i=0;i<size;i++)
	{	
		std::cout << "data[" << i << "] = "; 
		for(int j=0; j<dimension; j++)
			std::cout << data[i][j] << " ";
		std::cout << std::endl;
	}
*/
	//number of expected clusters
	int clusters = 2;

	movmf obj(dimension, clusters, size, data);


	obj.initialize();
	obj.run();
	obj.print();

	return 0;
}

